#include "hip/hip_runtime.h"
#include "tiny_gsplat.hpp"

#ifdef USE_HIP
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#else
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#endif

namespace cg = cooperative_groups;

namespace tinygsplat
{
// https://alexminnaar.com/2019/03/05/cuda-kmeans.html
__device__ float distanceCUDA(const float x1, const float x2)
{
	return sqrt((x2 - x1) * (x2 - x1));
}

// This function finds the centroid value based on the points that are
// classified as belonding to the respective class
__global__ void updateCentersCUDA(
	const float *values,
	const int *ids,
	float *centers,
	int *center_sizes,
	const int n_values,
	const int n_centers)
{
	auto idx = cg::this_grid().thread_rank();
	auto block = cg::this_thread_block();
	if (idx >= n_values)
		return;
	__shared__ float collected_values[256];
	collected_values[block.thread_rank()] = values[idx];

	__shared__ int collected_ids[256];
	collected_ids[block.thread_rank()] = ids[idx];

	block.sync();

	// One thread per block take on the task to gather the values
	if (block.thread_rank() == 0)
	{
		float block_center_sums[256] = {0};
		int block_center_sizes[256] = {0};
		for (int i = 0; i < 256 && idx + i < n_values; ++i)
		{
			int clust_id = collected_ids[i];
			block_center_sums[clust_id] += collected_values[i];
			block_center_sizes[clust_id] += 1;
		}

		for (int i = 0; i < n_centers; ++i)
		{
			atomicAdd(&centers[i], block_center_sums[i]);
			atomicAdd(&center_sizes[i], block_center_sizes[i]);
		}
	}
}

void updateCenters(
	const float *values,
	const int *ids,
	float *centers,
	int *center_sizes,
	const int n_values,
	const int n_centers)
{
	updateCentersCUDA<<<(n_values + 255) / 256, 256>>>(
		values,
		ids,
		centers,
		center_sizes,
		n_values,
		n_centers);
}

// This function finds the closest centroid for each point
__global__ void updateIdsCUDA(
	const float *values,
	int *ids,
	const float *centers,
	const int n_values,
	const int n_centers)
{
	auto idx = cg::this_grid().thread_rank();
	auto block = cg::this_thread_block();

	if (idx >= n_values)
		return;

	float min_dist = INFINITY;
	int closest_centroid = 0;

	__shared__ float collected_centers[256];

	block.sync();
	collected_centers[block.thread_rank()] = centers[block.thread_rank()];
	block.sync();

	for (int i = 0; i < n_centers; ++i)
	{
		float dist = distanceCUDA(values[idx], collected_centers[i]);

		if (dist < min_dist)
		{
			min_dist = dist;
			closest_centroid = i;
		}
	}

	ids[idx] = closest_centroid;
}

void updateIds(
	const float *values,
	int *ids,
	const float *centers,
	const int n_values,
	const int n_centers)
{
	updateIdsCUDA<<<(n_values + 255) / 256, 256>>>(
		values,
		ids,
		centers,
		n_values,
		n_centers);
}

// Works with 256 centers 1 dimensional data only
std::tuple<std::vector<int>, std::vector<float>>  kmeans_cluster(
	const std::vector<float>& values,
	const std::vector<float>& centers,
	const float tol,
	const int max_iterations)
{
	const int n_values = values.size();
	const int n_centers = centers.size();
	std::vector<int> ids(n_values,0);
	std::vector<float> new_centers(n_centers, 0.0f);
	std::vector<float> old_centers(n_centers, 0.0f);
	std::vector<int> center_sizes(n_centers, 0);
	new_centers = centers;
	float* d_centers, *d_old_centers, *d_values;
	int* d_center_sizes,*d_ids;
	hipMalloc(&d_centers,sizeof(float) * n_centers);
	hipMalloc(&d_old_centers, sizeof(float) * n_centers);
	hipMalloc(&d_center_sizes, sizeof(int) * n_values);
	hipMemcpy(d_center_sizes, center_sizes.data(), sizeof(int) * n_centers, hipMemcpyHostToDevice);

	hipMalloc(&d_values, sizeof(float) * n_values);
	hipMalloc(&d_ids, sizeof(int) * n_values);
	hipMemcpy(d_values, values.data(), sizeof(float) * n_values, hipMemcpyHostToDevice);
	hipMemcpy(d_ids, ids.data(), sizeof(int) * n_values, hipMemcpyHostToDevice);
	for (int i = 0; i < max_iterations; ++i)
	{
		hipMemcpy(d_centers, new_centers.data(), sizeof(float) * n_centers, hipMemcpyHostToDevice);
		hipMemcpy(d_old_centers,old_centers.data(), sizeof(float) * n_centers, hipMemcpyHostToDevice);

		updateIds(
			d_values,
			d_ids,
			d_centers,
			n_values,
			n_centers);
		
		hipMemcpy(d_old_centers,d_centers, sizeof(float) * n_centers,hipMemcpyDeviceToDevice);
		hipMemset(d_centers,0, sizeof(float) * n_centers);
		hipMemset(d_center_sizes, 0, sizeof(float) * n_centers);

		updateCenters(
			d_values,
			d_ids,
			d_centers,
			d_center_sizes,
			n_values,
			n_centers);

		hipMemcpy(new_centers.data(), d_centers, sizeof(float) * n_centers, hipMemcpyDeviceToHost);
		hipMemcpy(old_centers.data(), d_old_centers, sizeof(float) * n_centers, hipMemcpyDeviceToHost);
		hipMemcpy(center_sizes.data(), d_center_sizes, sizeof(int) * n_centers, hipMemcpyDeviceToHost);
		float center_shift = 0;
		for (auto j = 0; j < n_centers; j++)
		{
			if (center_sizes[j] != 0)
			{
				new_centers[j] /= center_sizes[j];
				center_shift += std::abs(old_centers[j] - new_centers[j]);
			}
			else 
				new_centers[j] = 0;
		}
		if (center_shift < tol)
			break;
	}
	hipMemcpy(d_centers, new_centers.data(), sizeof(float) * n_centers, hipMemcpyHostToDevice);
	updateIds(
		d_values,
		d_ids,
		d_centers,
		n_values,
		n_centers);
	hipMemcpy(new_centers.data(), d_centers, sizeof(float) * n_centers, hipMemcpyDeviceToHost);
	hipMemcpy(ids.data(), d_ids, sizeof(int) * n_values, hipMemcpyDeviceToHost);
	hipFree(d_centers);
	hipFree(d_old_centers);
	hipFree(d_center_sizes);
	hipFree(d_values);
	hipFree(d_ids);
	return std::make_tuple(ids, new_centers);
}

} // namespace tinygsplat